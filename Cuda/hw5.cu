
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <ctype.h>
#include <sys/types.h>

//----------------------------------------------------------------------------//
//----------------------------------ppmFile.c----------------------------------//
//----------------------------------------------------------------------------//

typedef struct Image
{
	  int width;
	  int height;
	  unsigned char *data;
} Image;

/************************ private functions ****************************/

/* die gracelessly */

static void
die(char const *message)
{
    fprintf(stderr, "ppm: %s\n", message);
    exit(1);
}


/* check a dimension (width or height) from the image file for reasonability */

static void
checkDimension(int dim)
{
    if (dim < 1 || dim > 6000) 
    die("file contained unreasonable width or height");
}


	/* read a header: verify format and get width and height */

static void
readPPMHeader(FILE *fp, int *width, int *height)
{
    char ch;
    int  maxval;

    if (fscanf(fp, "P%c\n", &ch) != 1 || ch != '6') 
    die("file is not in ppm raw format; cannot read");

    /* skip comments */
    ch = getc(fp);
    while (ch == '#')
    {
        do {
    ch = getc(fp);
        } while (ch != '\n');	/* read to the end of the line */
        ch = getc(fp);            
    }

    if (!isdigit(ch)) die("cannot read header information from ppm file");

    ungetc(ch, fp);		/* put that digit back */

    /* read the width, height, and maximum value for a pixel */
    fscanf(fp, "%d%d%d\n", width, height, &maxval);

    if (maxval != 255) die("image is not true-color (24 bit); read failed");
    
    checkDimension(*width);
    checkDimension(*height);
}

	/************************ exported functions ****************************/
__host__
Image *
ImageCreate(int width, int height)
{
    Image *image = (Image *) malloc(sizeof(Image));

    if (!image) die("cannot allocate memory for new image");

    image->width  = width;
    image->height = height;
    image->data   = (unsigned char *) malloc(width * height * 3);

    if (!image->data) die("cannot allocate memory for new image");

    return image;
}
	  
__host__
Image *
ImageRead(char const *filename)
{
    int width, height, num, size;
    //unsigned  *p;

    Image *image = (Image *) malloc(sizeof(Image));
    FILE  *fp    = fopen(filename, "rb");

    if (!image) die("cannot allocate memory for new image");
    if (!fp)    die("cannot open file for reading");

    readPPMHeader(fp, &width, &height);

    size          = width * height * 3;
    image->data   = (unsigned  char*) malloc(size);
    image->width  = width;
    image->height = height;

    if (!image->data) die("cannot allocate memory for new image");

    num = fread((void *) image->data, 1, (size_t) size, fp);

    if (num != size) die("cannot read image data from file");

    fclose(fp);

    return image;
}

__host__
void ImageWrite(Image *image, char const *filename)
{
    int num;
    int size = image->width * image->height * 3;

    FILE *fp = fopen(filename, "wb");

    if (!fp) die("cannot open file for writing");

    fprintf(fp, "P6\n%d %d\n%d\n", image->width, image->height, 255);

    num = fwrite((void *) image->data, 1, (size_t) size, fp);

    if (num != size) die("cannot write image data to file");

    fclose(fp);
}  

__host__
int
ImageWidth(Image *image)
{
    return image->width;
}

__host__
int
ImageHeight(Image *image)
{
    return image->height;
}

__host__
void   
ImageClear(Image *image, unsigned char red, unsigned char green, unsigned char blue)
{
    int i;
    int pix = image->width * image->height;

    unsigned char *data = image->data;

    for (i = 0; i < pix; i++)
    {
        *data++ = red;
        *data++ = green;
        *data++ = blue;
    }
}

__device__
void ImageSetPixel(unsigned char* data, int x, int y, int chan, unsigned char val,int width)    // changed for data use
{
    int offset = (y * width + x) * 3 + chan;

    data[offset] = val;
}


__device__
unsigned  char ImageGetPixel(unsigned char* data, int x, int y, int chan, int width)    //changed for data use
{
    int offset = (y * width + x) * 3 + chan;

    return data[offset];
}    


//========================================================================================//
//==============================          MY CODE          ===============================//
//========================================================================================//

typedef struct pix{
    int r,g,b;

}pix;


//--------------------------------KERNEL FUNCTION---------------------//   

__device__
pix getAvg(unsigned char* data,int w,int h,int r,int x, int y){
    pix avg = {0};

    
    int xMin, xMax, yMin, yMax;

    if((xMin = x-r) < 0) xMin = 0;
    if((yMin = y-r) < 0) yMin = 0;              //define bounds
    if((xMax = x+r) > w-1) xMax = w;
    if((yMax = y+r) > h-1) yMax = h;

    int i;
    int j;
        
    for(i=yMin; i < yMax; i++){        //i is y, j is x for row first iteration
        for(j=xMin; j < xMax; j++){    //efficient for cache
        
        avg.r += ImageGetPixel(data,j,i,0,w); 
        avg.g += ImageGetPixel(data,j,i,1,w); 
        avg.b += ImageGetPixel(data,j,i,2,w); 

        }
    }


    int num = (yMax-yMin)*(xMax-xMin);
    

    avg.r = avg.r/num;
    avg.g = avg.g/num;
    avg.b = avg.b/num;

    return avg;    
    
}

//--------------------------KERNEL---------------------------//

__global__
void myKernel(unsigned char* oldData, unsigned char* newData,int WIDTH, int HEIGHT, int r){


    int indx = blockIdx.x * blockDim.x + threadIdx.x;
    int indy = blockIdx.y * blockDim.y + threadIdx.y;

    pix avg;

    int stride_x = gridDim.x*blockDim.x;
    int stride_y = gridDim.y*blockDim.y;

    int i;
    int j;
    for(i=indy; i<HEIGHT; i+= stride_y){
        for(j=indx; j<WIDTH; j+= stride_x){

            avg = getAvg(oldData,WIDTH,HEIGHT,r,j,i);


            ImageSetPixel(newData,j,i,0,avg.r,WIDTH);
            ImageSetPixel(newData,j,i,1,avg.g,WIDTH);
            ImageSetPixel(newData,j,i,2,avg.b,WIDTH);
        }
    }
}


//------------------------------------MAIN----------------------------------//

int main(int argc, char *argv[]){

    //--------------Handle Input Arguments
    int r = atoi(argv[1]);
    char const * inFile = argv[2];
    char const * outFile = argv[3];

    //--------------Create
    Image* oldPic;
    Image* newPic;
    
    oldPic = ImageRead(inFile);                  //read old

    int WIDTH = ImageWidth(oldPic);
    int HEIGHT = ImageHeight(oldPic);

    newPic = ImageCreate(WIDTH,HEIGHT);        //make new same size as old

    printf("Processing...\n");

    //------------------cuda init----------------//

    dim3 blockDim(32,32);   //1024
    dim3 gridDim(20,20);

    int dsize = WIDTH*HEIGHT*3;         //size of data

    unsigned char* oldDataDevice;             //device data 
    unsigned char* newDataDevice;

    hipMalloc(&oldDataDevice,dsize);
    hipMalloc(&newDataDevice,dsize);

    hipMemcpy(oldDataDevice,oldPic->data,dsize,hipMemcpyHostToDevice);    //copy to device data

    //----------------------KERNEL--------------------//
    
    myKernel<<<gridDim,blockDim>>>(oldDataDevice,newDataDevice,WIDTH,HEIGHT,r);
    hipDeviceSynchronize();

    //----------------------post proccess------------------------//

    hipMemcpy(newPic->data,newDataDevice,dsize, hipMemcpyDeviceToHost);           //copy back
    ImageWrite(newPic,outFile);

    hipFree(oldDataDevice);
    hipFree(newDataDevice);
    printf("New picture written to: %s\n",outFile);

    return 0;
}    

